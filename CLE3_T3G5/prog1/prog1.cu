#include "hip/hip_runtime.h"
/**
 *   Tomás Oliveira e Silva, November 2017
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

#ifndef SECTOR_SIZE
	#define SECTOR_SIZE 512
#endif
#ifndef N_SECTORS
	#define N_SECTORS (1 << 21) // it can go as high as (1 << 21)
#endif

#ifndef N 1024
	#define N 1024
#endif

// allusion to internal functions

static bool readIntegerSequence(int** integerSequence, int* sequenceLen, char* fileName);

static void validateArray(int** integerSequence, int* sequenceLen);

__global__ static void sort_sequence_cuda_kernel(int * __restrict__ integerSequence, int subSequenceLen, int startOffset, int endOffset);

static double get_delta_time(void);

/**
 *   main program
 */
int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
    if (sizeof(unsigned int) != (size_t)4)
        return 1; // it fails with prejudice if an integer does not have 4 bytes
	
	// verify arguments number
    if (argc != 2)
    {
        printf("Wrong number of arguments!\n");
        return 1;
    }

    // set up the device

    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
	
	// parse integer sequence in host memory
	
	int *integerSequence = NULL;
	int sequenceLen = 0;
	
	if (!readIntegerSequence(&integerSequence, &sequenceLen, argv[1]))
	{
		return 1;
	}
	
	// create memory areas in device memory
	
	int *deviceIntegerSequence;

	if (sequenceLen > (size_t)5e9)
	{
		fprintf(stderr, "The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
		exit(1);
	}

	CHECK(hipMalloc((void **)&deviceIntegerSequence, sequenceLen));

    // copy the host data to the device memory

    CHECK(hipMemcpy(deviceIntegerSequence, integerSequence, sequenceLen, hipMemcpyHostToDevice));

    // run the computational kernel

	(void)get_delta_time();

	for (int iter = 0; iter < 9; iter++)
	{
		unsigned int gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ;
		
		blockDimX = 1 << 0; // optimize!
		blockDimY = 1 << 0; // optimize!
		blockDimZ = 1 << 0; // do not change!
		gridDimX = 1 << 21; // optimize!
		gridDimY = 1 << 0;  // optimize!
		gridDimZ = 1 << 0;  // do not change!
		
		switch (iter)
		{
			case 0:
			
				break;
			case 1:
				
				break;
			case 2:
				
				break;
			case 3:
				
				break;
			case 4:
				
				break;
			case 5:
				
				break;
			case 6:
				
				break;
			case 7:
				
				break;
			case 8:
				
				break;
			case 9:
				
				break;
			default:
				break;
		}
		
		dim3 grid(gridDimX, gridDimY, gridDimZ);
		dim3 block(blockDimX, blockDimY, blockDimZ);
		
		sort_sequence_cuda_kernel<<<grid, block>>>();	// sort sequence
		CHECK(hipDeviceSynchronize()); 				// wait for kernel to finish
		CHECK(hipGetLastError());      				// check for kernel errors
	}
	
	printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
							   gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time());

    // copy kernel result back to host side

    CHECK(hipMemcpy(integerSequence, deviceIntegerSequence, sequenceLen, hipMemcpyDeviceToHost));

    // free device global memory

    CHECK(hipFree(deviceIntegerSequence));

    // reset the device

    CHECK(hipDeviceReset());

	// validate results
	
	validateArray(&integerSequence, &sequenceLen);

    // free host memory

    free(integerSequence);

    return 0;
}

static bool readIntegerSequence(int** integerSequence, int* sequenceLen, char* fileName)
{
	FILE* filePointer = NULL;
	// open binary file
	if ((filePointer = fopen(fileName, "rb")) == NULL)
	{
		fprintf(stderr, "error on opening file \"%s\"\n", fileName);
		return false;
	}
	
	// get number of sequence elements
    if (fread(sequenceLen, sizeof(int), 1, filePointer) == EOF)
	{
		fprintf(stderr, "error on reading integer sequence length\n");
		return false;
	}
	
	// alocate integer sequence memory
	if ((*integerSequence = malloc((*sequenceLen) * sizeof(int))) == NULL)
	{
		fprintf(stderr, "error on allocating space to file name\n");
		return false;
	}

    // get the sequence of integers
    for (int i = 0; i < (*sequenceLen); i++)
    {	
		if (fread((*integerSequence) + i, sizeof(int), 1, filePointer) == EOF)
		{
			fprintf(stderr, "error on reading integer sequence length\n");
			return false;
		}
    }
	
	printf("Integer sequence parsed\n");
	
	// close binary file
	if (fclose(filePointer) == EOF)
	{
		fprintf(stderr, "error on closing text file \"%s\"\n", fileName);
		return false;
	}
	
	return true;
}

static void validateArray(int** integerSequence, int* sequenceLen)
{
    for (int i = 0; i < *sequenceLen - 1; i++)
    {
        if ((*integerSequence)[i] > (*integerSequence)[i + 1])
        {
            printf("Error in position %d between element %d and %d\n", i, (*integerSequence)[i], (*integerSequence)[i + 1]);
            return;
        }
    }
    printf("Everything is OK!\n");
}

__global__ static void sort_sequence_cuda_kernel(int * __restrict__ integerSequence, int subSequenceLen, int startOffset, int endOffset)
{
    unsigned int x, y, idx;

    // compute the thread number

    x = (unsigned int)threadIdx.x + (unsigned int)blockDim.x * (unsigned int)blockIdx.x;
    y = (unsigned int)threadIdx.y + (unsigned int)blockDim.y * (unsigned int)blockIdx.y;
    idx = (unsigned int)blockDim.x * (unsigned int)gridDim.x * y + x;

    // sort sequence
	
	for (int k = 2; k <= *subSequenceLen; k *= 2) // k is doubled every iteration
	{
		for (int j = k / 2; j > 0; j /= 2) // j is halved at every iteration, with truncation of fractional parts
		{
			for (int i = startOffset; i < endOffset; i++)
			{
				int l = i ^ j;
				if (l > i)
				{
					if ((((i & k) == 0) && (integerSequence[i] > integerSequence[l])) || (((i & k) != 0) && (integerSequence[i] < integerSequence[l])))
					{
						int temp = integerSequence[i];
						integerSequence[i] = integerSequence[l];
						integerSequence[l] = temp;
					}
				}
			}
		}
	}
	
	
}

static double get_delta_time(void)
{
    static struct timespec t0, t1;

    t0 = t1;
    if (clock_gettime(CLOCK_MONOTONIC, &t1) != 0)
    {
        perror("clock_gettime");
        exit(1);
    }
    return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
