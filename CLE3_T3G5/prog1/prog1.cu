#include "hip/hip_runtime.h"
/**
 *   Author Name, May 2023
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

#ifndef N
	#define N 1024
#endif

// allusion to internal functions

static bool readIntegerSequence(int** integerSequence, int* sequenceLen, char* fileName);

static void validateArray(int** integerSequence, int* sequenceLen);

__global__ static void sort_sequence_cuda_kernel(int * __restrict__ integerSequence, int iter, int subSequenceLen);

static double get_delta_time(void);

/**
 *   main program
 */
int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
    if (sizeof(unsigned int) != (size_t)4)
        return 1; // it fails with prejudice if an integer does not have 4 bytes
	
	// verify arguments number
    if (argc != 2)
    {
        printf("Wrong number of arguments!\n");
        return 1;
    }

    // set up the device

    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
	
	// parse integer sequence in host memory
	
	int *integerSequence = NULL;
	int sequenceLen = 0;
	
	if (!readIntegerSequence(&integerSequence, &sequenceLen, argv[1]))
	{
		return 1;
	}
	
	//printf("Integer sequence length: %d\n", sequenceLen);
	
	// create memory area in device memory
	
	int *deviceIntegerSequence = NULL;

	if (sequenceLen > (size_t)5e9)
	{
		fprintf(stderr, "The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
		exit(1);
	}

	CHECK(hipMalloc((void **)&deviceIntegerSequence, sequenceLen * sizeof(int)));

    // copy the host data to the device memory

    CHECK(hipMemcpy(deviceIntegerSequence, integerSequence, sequenceLen * sizeof(int), hipMemcpyHostToDevice));

    // run the computational kernel
	
	unsigned int gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ;
	
	blockDimX = 1 << 0; // optimize!
	blockDimY = 1 << 0; // optimize!
	blockDimZ = 1 << 0; // do not change!
	gridDimX = 1 << 0; 	// optimize!
	gridDimY = 1 << 0;  // optimize!
	gridDimZ = 1 << 0;  // do not change!

	(void)get_delta_time();
	
	for (int iter = 0; iter <= 10; iter++)
	{
		printf("Iteration = %d\n", iter);
		
		switch (iter)
		{
			case 0:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 10;	// optimize!
				break;
			case 1:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 9; 	// optimize!
				break;
			case 2:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 8; 	// optimize!
				break;
			case 3:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 7; 	// optimize!
				break;
			case 4:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 6; 	// optimize!
				break;
			case 5:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 5; 	// optimize!
				break;
			case 6:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 4; 	// optimize!
				break;
			case 7:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 3; 	// optimize!
				break;
			case 8:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 2; 	// optimize!
				break;
			case 9:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 1; 	// optimize!
				break;
			case 10:
				blockDimX = 1 << 0; // optimize!
				gridDimX = 1 << 0; 	// optimize!
				break;
			default:
				break;
		}
		
		dim3 grid(gridDimX, gridDimY, gridDimZ);
		dim3 block(blockDimX, blockDimY, blockDimZ);
		
		sort_sequence_cuda_kernel<<<grid, block>>>(deviceIntegerSequence, iter, (1 << iter) * N);	// sort sequence
		CHECK(hipDeviceSynchronize()); 				// wait for kernel to finish
		CHECK(hipGetLastError());      				// check for kernel errors
	}
	
	printf("\nElapsed time = %.6f s\n", get_delta_time());

    // copy kernel result back to host side

    CHECK(hipMemcpy(integerSequence, deviceIntegerSequence, sequenceLen * sizeof(int), hipMemcpyDeviceToHost));

    // free device global memory

    CHECK(hipFree(deviceIntegerSequence));

    // reset the device

    CHECK(hipDeviceReset());

	// validate results
	
	validateArray(&integerSequence, &sequenceLen);

    // free host memory

    free(integerSequence);

    return 0;
}

static bool readIntegerSequence(int** integerSequence, int* sequenceLen, char* fileName)
{
	FILE* filePointer = NULL;
	// open binary file
	if ((filePointer = fopen(fileName, "rb")) == NULL)
	{
		fprintf(stderr, "error on opening file \"%s\"\n", fileName);
		return false;
	}
	
	// get number of sequence elements
    if (fread(sequenceLen, sizeof(int), 1, filePointer) == EOF)
	{
		fprintf(stderr, "error on reading integer sequence length\n");
		return false;
	}
	
	// alocate integer sequence memory
	if ((*integerSequence = (int *)malloc((*sequenceLen) * sizeof(int))) == NULL)
	{
		fprintf(stderr, "error on allocating space to file name\n");
		return false;
	}

    // get the sequence of integers
    for (int i = 0; i < (*sequenceLen); i++)
    {	
		if (fread((*integerSequence) + i, sizeof(int), 1, filePointer) == EOF)
		{
			fprintf(stderr, "error on reading integer sequence length\n");
			return false;
		}
    }
	
	printf("Integer sequence parsed\n");
	
	// close binary file
	if (fclose(filePointer) == EOF)
	{
		fprintf(stderr, "error on closing text file \"%s\"\n", fileName);
		return false;
	}
	
	return true;
}

static void validateArray(int** integerSequence, int* sequenceLen)
{
    for (int i = 0; i < *sequenceLen - 1; i++)
    {
        if ((*integerSequence)[i] > (*integerSequence)[i + 1])
        {
            printf("Error in position %d between element %d and %d\n", i, (*integerSequence)[i], (*integerSequence)[i + 1]);
            return;
        }
    }
    printf("Everything is OK!\n");
}

__global__ static void sort_sequence_cuda_kernel(int * __restrict__ integerSequence, int iter, int subSequenceLen)
{
    unsigned int x, y, idx;

    // compute the thread number

    x = (unsigned int)threadIdx.x + (unsigned int)blockDim.x * (unsigned int)blockIdx.x;
    y = (unsigned int)threadIdx.y + (unsigned int)blockDim.y * (unsigned int)blockIdx.y;
    idx = (unsigned int)blockDim.x * (unsigned int)gridDim.x * y + x;
	
	//printf("THREAD IDX: %d\n", idx);
	
	// sort sequence
	if (subSequenceLen == N)
	{
		for (int k = 2; k <= subSequenceLen; k *= 2) // k is doubled every iteration
		{
			for (int j = k / 2; j > 0; j /= 2) // j is halved at every iteration, with truncation of fractional parts
			{
				for (int i = 0; i < (1 << iter) * N; i++)
				{
					int m = N * (1 << iter) * idx + i;
					int l = m ^ j;
					if (l > m)
					{
						if ((((m & k) == 0) && (integerSequence[m] > integerSequence[l])) || (((m & k) != 0) && (integerSequence[m] < integerSequence[l])))
						{
							int temp = integerSequence[m];
							integerSequence[m] = integerSequence[l];
							integerSequence[l] = temp;
						}
					}
				}
			}
		}
	}
	else
	{
		int k = subSequenceLen;
		
		for (int j = k / 2; j > 0; j /= 2) // j is halved at every iteration, with truncation of fractional parts
		{
			for (int i = 0; i < (1 << iter) * N; i++)
			{
				int m = N * (1 << iter) * idx + i;
				int l = m ^ j;
				if (l > m)
				{
					if ((((m & k) == 0) && (integerSequence[m] > integerSequence[l])) || (((m & k) != 0) && (integerSequence[m] < integerSequence[l])))
					{
						int temp = integerSequence[m];
						integerSequence[m] = integerSequence[l];
						integerSequence[l] = temp;
					}
				}
			}
		}
	}
}

static double get_delta_time(void)
{
    static struct timespec t0, t1;

    t0 = t1;
    if (clock_gettime(CLOCK_MONOTONIC, &t1) != 0)
    {
        perror("clock_gettime");
        exit(1);
    }
    return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
